// flame/flame-core/cuda/narrow_strided_backward.cu
// Scatter-add for narrow backward with dtype-correct atomic adds.

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <hip/hip_fp16.h>

enum DTypeTag { F32 = 0, F16 = 1, BF16 = 2, I32 = 3 };

static __device__ __forceinline__ void linear_to_indices(
    int64_t lin, const int64_t* __restrict__ shape, int rank, int64_t* __restrict__ idx)
{
    for (int i = rank - 1; i >= 0; --i) {
        int64_t dim = shape[i];
        idx[i] = lin % dim;
        lin /= dim;
    }
}

// Atomic add for BF16 using CAS on 32-bit word
static __device__ inline void atomicAdd_bf16(uint16_t* addr_b16, uint16_t val_b16)
{
    uintptr_t byte_addr = reinterpret_cast<uintptr_t>(addr_b16);
    uintptr_t aligned = byte_addr & ~0x3ULL; // 32-bit align
    bool high = (byte_addr & 0x2ULL) != 0ULL;
    unsigned int* base = reinterpret_cast<unsigned int*>(aligned);
    unsigned int old = *base;
    unsigned int assumed;
    do {
        assumed = old;
        uint16_t cur_b16 = high ? static_cast<uint16_t>((assumed >> 16) & 0xFFFF) : static_cast<uint16_t>(assumed & 0xFFFF);
        // bf16 -> float
        uint32_t cur_bits = static_cast<uint32_t>(cur_b16) << 16;
        float cur = __int_as_float(cur_bits);
        // add
        uint32_t add_bits = static_cast<uint32_t>(val_b16) << 16;
        float addf = __int_as_float(add_bits);
        float sum = cur + addf;
        // float -> bf16 (round to nearest even approx)
        uint32_t sum_bits = __float_as_int(sum);
        uint16_t new_b16 = static_cast<uint16_t>((sum_bits + 0x8000u) >> 16);
        unsigned int new_word;
        if (high) {
            new_word = (assumed & 0x0000FFFFu) | (static_cast<unsigned int>(new_b16) << 16);
        } else {
            new_word = (assumed & 0xFFFF0000u) | static_cast<unsigned int>(new_b16);
        }
        old = atomicCAS(base, assumed, new_word);
    } while (old != assumed);
}

// Atomic add for FP16 using native atomicAdd if available, else CAS
static __device__ inline void atomicAdd_f16(uint16_t* addr_h, uint16_t val_h)
{
#if __CUDA_ARCH__ >= 700
    __half* hp = reinterpret_cast<__half*>(addr_h);
    __half addv = *reinterpret_cast<__half*>(&val_h);
    atomicAdd(hp, addv);
#else
    uintptr_t byte_addr = reinterpret_cast<uintptr_t>(addr_h);
    uintptr_t aligned = byte_addr & ~0x3ULL;
    bool high = (byte_addr & 0x2ULL) != 0ULL;
    unsigned int* base = reinterpret_cast<unsigned int*>(aligned);
    unsigned int old = *base;
    unsigned int assumed;
    do {
        assumed = old;
        uint16_t cur_h = high ? static_cast<uint16_t>((assumed >> 16) & 0xFFFF) : static_cast<uint16_t>(assumed & 0xFFFF);
        __half cur = *reinterpret_cast<__half*>(&cur_h);
        __half addv = *reinterpret_cast<__half*>(&val_h);
        float sumf = __half2float(cur) + __half2float(addv);
        __half sumh = __float2half(sumf);
        uint16_t new_h = *reinterpret_cast<uint16_t*>(&sumh);
        unsigned int new_word;
        if (high) {
            new_word = (assumed & 0x0000FFFFu) | (static_cast<unsigned int>(new_h) << 16);
        } else {
            new_word = (assumed & 0xFFFF0000u) | static_cast<unsigned int>(new_h);
        }
        old = atomicCAS(base, assumed, new_word);
    } while (old != assumed);
#endif
}

extern "C" __global__
void narrow_backward_scatter_add_kernel(
    const uint8_t* __restrict__ grad_out,
    uint8_t* __restrict__ grad_in,
    int rank,
    const int64_t* __restrict__ out_shape,
    const int64_t* __restrict__ in_strides,
    const int64_t* __restrict__ out_strides,
    int dim,
    int64_t start,
    int64_t elem_size,
    int dtype_tag,
    int64_t n_elements)
{
    int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n_elements) return;

    // Out multi-index
    int64_t idx_buf[8];
    linear_to_indices(tid, out_shape, rank, idx_buf);

    // Input offset in elements
    int64_t in_off_e = 0;
    for (int i = 0; i < rank; ++i) {
        int64_t idx_i = idx_buf[i];
        if (i == dim) idx_i += start;
        in_off_e += idx_i * in_strides[i];
    }

    const uint8_t* s = grad_out + tid * elem_size;
    uint8_t* d = grad_in + in_off_e * elem_size;

    switch (dtype_tag) {
    case F32: {
        float* dp = reinterpret_cast<float*>(d);
        const float* sp = reinterpret_cast<const float*>(s);
        atomicAdd(dp, *sp);
        break;
    }
    case I32: {
        int* dp = reinterpret_cast<int*>(d);
        const int* sp = reinterpret_cast<const int*>(s);
        atomicAdd(dp, *sp);
        break;
    }
    case F16: {
        uint16_t* dp = reinterpret_cast<uint16_t*>(d);
        const uint16_t* sp = reinterpret_cast<const uint16_t*>(s);
        atomicAdd_f16(dp, *sp);
        break;
    }
    case BF16: {
        uint16_t* dp = reinterpret_cast<uint16_t*>(d);
        const uint16_t* sp = reinterpret_cast<const uint16_t*>(s);
        atomicAdd_bf16(dp, *sp);
        break;
    }
    default: {
        // Fallback: plain byte-wise copy (safe for narrow scatter – no overlap)
        for (int64_t i = 0; i < elem_size; ++i) {
            reinterpret_cast<uint8_t*>(d)[i] = reinterpret_cast<const uint8_t*>(s)[i];
        }
        break;
    }
    }
}

extern "C" int flame_narrow_backward_scatter_add_launch(
    const void* grad_out,
    void* grad_in,
    int rank,
    const int64_t* out_shape_host,
    const int64_t* in_strides_host,
    const int64_t* out_strides_host,
    int dim,
    int64_t start,
    int64_t elem_size,
    int dtype_tag,
    int64_t n_elements,
    void* stream_void)
{
    hipStream_t stream = reinterpret_cast<hipStream_t>(stream_void);
    // Copy metadata to device
    int64_t *d_shape = nullptr, *d_in_strides = nullptr, *d_out_strides = nullptr;
    size_t meta_sz = sizeof(int64_t) * static_cast<size_t>(rank);
    if (hipMalloc(&d_shape, meta_sz) != hipSuccess) return (int)hipGetLastError();
    if (hipMalloc(&d_in_strides, meta_sz) != hipSuccess) { hipFree(d_shape); return (int)hipGetLastError(); }
    if (hipMalloc(&d_out_strides, meta_sz) != hipSuccess) { hipFree(d_shape); hipFree(d_in_strides); return (int)hipGetLastError(); }
    hipMemcpyAsync(d_shape, out_shape_host, meta_sz, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_in_strides, in_strides_host, meta_sz, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_out_strides, out_strides_host, meta_sz, hipMemcpyHostToDevice, stream);

    int threads = 256;
    int blocks = (int)((n_elements + threads - 1) / threads);
    narrow_backward_scatter_add_kernel<<<blocks, threads, 0, stream>>>(
        (const uint8_t*)grad_out, (uint8_t*)grad_in,
        rank, d_shape, d_in_strides, d_out_strides,
        dim, start, elem_size, dtype_tag, n_elements
    );

    hipError_t err = hipGetLastError();
    hipFree(d_shape);
    hipFree(d_in_strides);
    hipFree(d_out_strides);
    return (err == hipSuccess) ? 0 : (int)err;
}
