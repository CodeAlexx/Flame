// flame/flame-core/cuda/narrow_strided.cu
// General byte-wise narrow gather for any dimension with strided input.

#include <hip/hip_runtime.h>
#include <stdint.h>

static __device__ __forceinline__ void linear_to_indices(
    int64_t lin, const int64_t* __restrict__ shape, int rank, int64_t* __restrict__ idx)
{
    // Row-major unravel: idx[0]..idx[rank-1]
    for (int i = rank - 1; i >= 0; --i) {
        int64_t dim = shape[i];
        idx[i] = lin % dim;
        lin /= dim;
    }
}

extern "C" __global__
void narrow_strided_kernel(
    const uint8_t* __restrict__ src,
    uint8_t* __restrict__ dst,
    int rank,
    const int64_t* __restrict__ out_shape,     // [rank]
    const int64_t* __restrict__ src_strides,   // [rank], in elements
    const int64_t* __restrict__ out_strides,   // [rank], in elements (row-major)
    int dim,
    int64_t start,
    int64_t elem_size,
    int64_t n_elements)
{
    int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n_elements) return;

    // Compute output multi-index
    int64_t idx_buf[8]; // supports rank <= 8
    linear_to_indices(tid, out_shape, rank, idx_buf);

    // Map to source multi-index (add start on the narrowed dim)
    int64_t src_offset_elems = 0;
    for (int i = 0; i < rank; ++i) {
        int64_t idx_i = idx_buf[i];
        if (i == dim) idx_i += start;
        src_offset_elems += idx_i * src_strides[i];
    }

    // Byte addresses
    const uint8_t* s = src + src_offset_elems * elem_size;
    uint8_t* d = dst + tid * elem_size;

    // Copy elem_size bytes
    int64_t n8 = elem_size / 8;
    int64_t rem = elem_size % 8;
    const uint64_t* s64 = reinterpret_cast<const uint64_t*>(s);
    uint64_t* d64 = reinterpret_cast<uint64_t*>(d);
#pragma unroll
    for (int64_t i = 0; i < n8; ++i) d64[i] = s64[i];
    for (int64_t i = 0; i < rem; ++i) d[n8 * 8 + i] = s[n8 * 8 + i];
}

extern "C" int flame_narrow_strided_launch(
    const void* src,
    void* dst,
    int rank,
    const int64_t* out_shape_host,
    const int64_t* src_strides_host,
    const int64_t* out_strides_host,
    int dim,
    int64_t start,
    int64_t elem_size,
    int64_t n_elements,
    void* stream_void)
{
    hipStream_t stream = reinterpret_cast<hipStream_t>(stream_void);

    // Copy tiny metadata arrays to device
    int64_t *d_shape = nullptr, *d_src_strides = nullptr, *d_out_strides = nullptr;
    size_t meta_sz = sizeof(int64_t) * static_cast<size_t>(rank);
    if (hipMalloc(&d_shape, meta_sz) != hipSuccess) return (int)hipGetLastError();
    if (hipMalloc(&d_src_strides, meta_sz) != hipSuccess) { hipFree(d_shape); return (int)hipGetLastError(); }
    if (hipMalloc(&d_out_strides, meta_sz) != hipSuccess) { hipFree(d_shape); hipFree(d_src_strides); return (int)hipGetLastError(); }
    hipMemcpyAsync(d_shape, out_shape_host, meta_sz, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_src_strides, src_strides_host, meta_sz, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_out_strides, out_strides_host, meta_sz, hipMemcpyHostToDevice, stream);

    int threads = 256;
    int blocks = (int)((n_elements + threads - 1) / threads);

    narrow_strided_kernel<<<blocks, threads, 0, stream>>>(
        (const uint8_t*)src, (uint8_t*)dst,
        rank, d_shape, d_src_strides, d_out_strides,
        dim, start, elem_size, n_elements
    );

    hipError_t err = hipGetLastError();
    hipFree(d_shape);
    hipFree(d_src_strides);
    hipFree(d_out_strides);
    return (err == hipSuccess) ? 0 : (int)err;
}
